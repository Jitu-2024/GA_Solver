#include "hip/hip_runtime.h"
#include "mutation.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <stdexcept>

// CUDA kernel for performing swap mutation
__global__ void swapMutationKernel(int* d_population, int numGenomes, int numCities, float mutationRate, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numGenomes) {
        hiprandState localState = states[idx];
        float randomValue = hiprand_uniform(&localState);

        // Perform mutation based on mutationRate
        if (randomValue < mutationRate) {
            // Randomly select two indices in the chromosome to swap
            int city1 = hiprand(&localState) % numCities;
            int city2 = hiprand(&localState) % numCities;

            // Ensure city1 and city2 are different
            while (city1 == city2) {
                city2 = hiprand(&localState) % numCities;
            }

            // Swap cities in the chromosome
            int temp = d_population[idx * numCities + city1];
            d_population[idx * numCities + city1] = d_population[idx * numCities + city2];
            d_population[idx * numCities + city2] = temp;
        }

        states[idx] = localState; // Save back the state
    }
}

// Initialize CUDA random states
__global__ void initializeRandomStates(hiprandState* states, unsigned long seed, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

// Constructor
Mutation::Mutation(int numCities, float mutationRate)
    : numCities(numCities), mutationRate(mutationRate), d_population(nullptr), d_mutatedPopulation(nullptr) {}

// Destructor
Mutation::~Mutation() {
    if (d_population) hipFree(d_population);
}

// Apply mutation using GPU
void Mutation::applyMutation(std::vector<std::vector<int>>& population) {
    int numGenomes = population.size();

    // Flatten the population into a 1D array for GPU processing
    std::vector<int> flatPopulation(numGenomes * numCities);
    for (int i = 0; i < numGenomes; ++i) {
        std::copy(population[i].begin(), population[i].end(), flatPopulation.begin() + i * numCities);
    }

    // Allocate memory on GPU
    hipMalloc(&d_population, flatPopulation.size() * sizeof(int));
    hipMemcpy(d_population, flatPopulation.data(), flatPopulation.size() * sizeof(int), hipMemcpyHostToDevice);

    // Allocate random states for each genome
    hiprandState* d_states;
    hipMalloc(&d_states, numGenomes * sizeof(hiprandState));
    initializeRandomStates<<<(numGenomes + 255) / 256, 256>>>(d_states, time(nullptr), numGenomes);

    // Launch kernel to perform swap mutation
    swapMutationKernel<<<(numGenomes + 255) / 256, 256>>>(d_population, numGenomes, numCities, mutationRate, d_states);

    // Copy back mutated population to host
    hipMemcpy(flatPopulation.data(), d_population, flatPopulation.size() * sizeof(int), hipMemcpyDeviceToHost);

    // Reshape flattened population back to 2D vector
    for (int i = 0; i < numGenomes; ++i) {
        std::copy(flatPopulation.begin() + i * numCities,
                  flatPopulation.begin() + (i + 1) * numCities,
                  population[i].begin());
    }

    // Free GPU memory
    hipFree(d_population);
    hipFree(d_states);
}
