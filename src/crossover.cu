#include "hip/hip_runtime.h"
// crossover.cu: Order crossover for TSPJ problem

#include "crossover.h"
#include "genome.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/random.h>

// Kernel for order crossover on GPU
__global__ void orderCrossoverKernel(const size_t* parent1, const size_t* parent2,
                                     size_t* child, size_t chromosomeLength,
                                     unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx == 0) { // Single thread selects crossover points
        thrust::default_random_engine rng(seed);
        thrust::uniform_int_distribution<size_t> dist(0, chromosomeLength - 1);

        size_t start = dist(rng);
        size_t end = dist(rng);

        if (start > end) {
            size_t temp = start;
            start = end;
            end = temp;
        }

        // Copy segment from parent1 to child
        for (size_t i = start; i <= end; ++i) {
            child[i] = parent1[i];
        }

        // Fill the rest from parent2, preserving order
        size_t childIdx = (end + 1) % chromosomeLength;
        for (size_t i = 0; i < chromosomeLength; ++i) {
            size_t candidate = parent2[(end + 1 + i) % chromosomeLength];

            // Check if candidate is already in the child
            bool exists = false;
            for (size_t j = start; j <= end; ++j) {
                if (child[j] == candidate) {
                    exists = true;
                    break;
                }
            }

            if (!exists) {
                child[childIdx] = candidate;
                childIdx = (childIdx + 1) % chromosomeLength;
            }
        }
    }
}

// Host function for order crossover
std::pair<Genome, Genome> performCrossover(const Genome& parent1, const Genome& parent2) {
    size_t chromosomeLength = parent1.citySequence.size();

    // Prepare device memory
    thrust::device_vector<size_t> d_parent1City(parent1.citySequence);
    thrust::device_vector<size_t> d_parent2City(parent2.citySequence);
    thrust::device_vector<size_t> d_childCity1(chromosomeLength);
    thrust::device_vector<size_t> d_childCity2(chromosomeLength);

    thrust::device_vector<size_t> d_parent1Job(parent1.jobSequence);
    thrust::device_vector<size_t> d_parent2Job(parent2.jobSequence);
    thrust::device_vector<size_t> d_childJob1(chromosomeLength);
    thrust::device_vector<size_t> d_childJob2(chromosomeLength);

    thrust::device_vector<size_t> d_parent1Pickup(parent1.pickupSequence);
    thrust::device_vector<size_t> d_parent2Pickup(parent2.pickupSequence);
    thrust::device_vector<size_t> d_childPickup1(chromosomeLength);
    thrust::device_vector<size_t> d_childPickup2(chromosomeLength);

    // Launch kernels for cities, jobs, and pickups
    int threadsPerBlock = 1; // Single thread selects crossover points
    int blocksPerGrid = 1;
    unsigned long seed = time(nullptr);

    orderCrossoverKernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(d_parent1City.data()),
        thrust::raw_pointer_cast(d_parent2City.data()),
        thrust::raw_pointer_cast(d_childCity1.data()),
        chromosomeLength, seed);

    orderCrossoverKernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(d_parent2City.data()),
        thrust::raw_pointer_cast(d_parent1City.data()),
        thrust::raw_pointer_cast(d_childCity2.data()),
        chromosomeLength, seed);

    orderCrossoverKernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(d_parent1Job.data()),
        thrust::raw_pointer_cast(d_parent2Job.data()),
        thrust::raw_pointer_cast(d_childJob1.data()),
        chromosomeLength, seed);

    orderCrossoverKernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(d_parent2Job.data()),
        thrust::raw_pointer_cast(d_parent1Job.data()),
        thrust::raw_pointer_cast(d_childJob2.data()),
        chromosomeLength, seed);

    orderCrossoverKernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(d_parent1Pickup.data()),
        thrust::raw_pointer_cast(d_parent2Pickup.data()),
        thrust::raw_pointer_cast(d_childPickup1.data()),
        chromosomeLength, seed);

    orderCrossoverKernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(d_parent2Pickup.data()),
        thrust::raw_pointer_cast(d_parent1Pickup.data()),
        thrust::raw_pointer_cast(d_childPickup2.data()),
        chromosomeLength, seed);

    // Copy results back to host
    Genome child1(chromosomeLength, chromosomeLength);
    Genome child2(chromosomeLength, chromosomeLength);

    thrust::copy(d_childCity1.begin(), d_childCity1.end(), child1.citySequence.begin());
    thrust::copy(d_childCity2.begin(), d_childCity2.end(), child2.citySequence.begin());

    thrust::copy(d_childJob1.begin(), d_childJob1.end(), child1.jobSequence.begin());
    thrust::copy(d_childJob2.begin(), d_childJob2.end(), child2.jobSequence.begin());

    thrust::copy(d_childPickup1.begin(), d_childPickup1.end(), child1.pickupSequence.begin());
    thrust::copy(d_childPickup2.begin(), d_childPickup2.end(), child2.pickupSequence.begin());

    return {child1, child2};
}
