#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "genetic_kernels.cuh"

// CUDA kernel for mutating genomes
__global__ void mutation_kernel(uint8_t* population, int total_bits, float mutation_rate, hiprandState* states) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Global thread index
    if (idx < total_bits) {
        // Generate a random number
        float rand_val = hiprand_uniform(&states[idx]);

        // Apply mutation based on mutation rate
        if (rand_val < mutation_rate) {
            population[idx / 8] ^= (1 << (idx % 8)); // Flip the bit
        }
    }
}

// Kernel for initializing random states
__global__ void init_random_states(hiprandState* states, unsigned long seed, int total_threads) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < total_threads) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

void gpu_mutate(uint8_t* population, int total_bits, float mutation_rate) {
    // Number of threads and blocks
    int threads_per_block = 256;
    int blocks = (total_bits + threads_per_block - 1) / threads_per_block;

    // Allocate memory for random states on GPU
    hiprandState* d_states;
    hipMalloc(&d_states, total_bits * sizeof(hiprandState));

    // Initialize random states
    init_random_states<<<blocks, threads_per_block>>>(d_states, 12345, total_bits);
    hipDeviceSynchronize();

    // Launch mutation kernel
    mutation_kernel<<<blocks, threads_per_block>>>(population, total_bits, mutation_rate, d_states);
    hipDeviceSynchronize();

    // Free random states memory
    hipFree(d_states);
}